#include "hvm.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <errno.h>

Port cmd(GNet* net, Port arg) {
    Str path = readback_str(net, book, arg);
    char* command = malloc(path.len);
    sprintf(command, "%s", path.buf);
    free(path.buf)

    FILE* pipe = popen(command, "r");
    if (pipe == NULL) {
        fprintf(stderr, "Command had no output or failed to run command '%s': %s\n", command, strerror(errno));
        return new_port(ERA, 0);
    }
    char buffer[512];
    Bytes output = { .buf = NULL, .len = 0 };
    while (fgets(buffer, sizeof(buffer), pipe) != NULL) {
        size_t len = strlen(buffer);
        char* new_output = realloc(output.buf, output.len + len + 1);
        if (new_output == NULL) {
            fprintf(stderr, "failed to allocate space for output of '%s': %s\n", command, strerror(errno));
            free(command);
            free(output.buf);
            pclose(pipe);
            return new_port(ERA, 0);
        }
        output.buf = new_output;
        strcpy(output.buf + output.len, buffer);
        output.len += len;
    }

    Port output_port = inject_bytes(net, &output);

    free(command);
    free(output.buf);
    pclose(pipe);
    return output_port;
}
