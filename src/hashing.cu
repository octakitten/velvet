#include "hip/hip_runtime.h"
// alright, here's our C function to create our pseudo random input values
// to reiterate, we need a hash function that will:
// - be roughly evenly distributed in a range from 0 to max(int24)
// - accept inputs from the domain 0 to max(uint24)
//
// there's another issue later on, which is that we will need another hashing function 
// to produce the vector arrays we need.
// the problem is that these arrays need to have only unique values.
// so the hashing function we use to generate them need to be one-to-one across the 
// domain and range we're interested in, probably just 0 to max(int243)
//
// this is more difficult and i dont know if multiplicative hashing works that way
//
// note: easy copy and paste from the c to cuda file here, just have to modify a couple things 

#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <errno.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "hvm.h"

#define BENDSIZEINT 8388608
#define BENDSIZEUINT 16777216

Port hashgrab(GNet* gnet, Port arg) {

    int amount = readback_bytes(gnet, arg);

    char* seeds = malloc(sizeof(char) * 3 * amount);
    FILE* seedfile = fopen("/dev/urandom", "r");
    fread(&seeds, sizeof(char) * 3 * amount, 1, seedfile);
    fclose(seedfile);
    
    char* int24array = malloc(sizeof(char) * 3 * amount);
    for (int i = 0; i < amount; i++) {
        int24array[i] = srand(seeds[i]);
    }

    free(seeds);
    return int24array;
}
